#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define UPPER 1000
#define LOWER 0

__global__
void MyPosition(int *x, int *y, int n)
{
  //Deixnei poios ein o arithmos tu thread
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  //printf("HALLO FROM THREAD : %d",idx);

if (idx<n){
	int my_num=x[idx];
    int my_place = 0;
     for (int i=0; i<n; i++)
		if ((my_num > x[i]) || ((my_num == x[i]) && (idx < i)))
			my_place++;
      __syncthreads();
     y[my_place] = my_num;
   }
}

int main(int argc, char *argv[])
{
   int *x_h, *y_h, *x_d, *y_d;
   int i;

   hipEvent_t total_start, total_stop, comp_start, comp_stop;
   float total_time, comp_time;
   hipEventCreate(&total_start);
   hipEventCreate(&total_stop);
   hipEventCreate(&comp_start);
   hipEventCreate(&comp_stop);

   if (argc != 3) {
		printf ("Usage : %s <array_size> <threads_per_block> \n", argv[0]);
		return 1;
   }

   const int n = strtol(argv[1], NULL, 10);
   const int threads_per_block = strtol(argv[2],NULL,10);
   //Memory on host
   x_h = ( int * ) malloc ( n * sizeof ( int ) );
   y_h = ( int * ) malloc ( n * sizeof ( int ) );

   if (x_h == NULL || y_h == NULL) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return 1;
    }

    // Allocate memory on device
    if (hipMalloc((void **)&x_d, n * sizeof(int)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory X.\n");
        return 1;
    }else if(hipMalloc((void **)&y_d, n * sizeof(int)) != hipSuccess){
      fprintf(stderr, "Failed to allocate device memory Y.\n");
        return 1;
    }

   //Memory on device
   hipMalloc((void **) &x_d,n * sizeof (int));
   hipMalloc((void **) &y_d,n * sizeof (int));


   for (i=0; i<n; i++)
		x_h[i] = n - i;
		//x[i] = (rand() % (UPPER - LOWER + 1)) + LOWER;

    //Set Num of blocks
    int blocks = (n+threads_per_block-1)/threads_per_block;

    // start total timing
    hipEventRecord(total_start);

    //Memory copy
    hipMemcpy( x_d, x_h, n*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( y_d, y_h, n*sizeof(int), hipMemcpyHostToDevice );

  //Start comp timing
    hipEventRecord(comp_start);

    printf("REACHED HERE \n");

    MyPosition<<<blocks,threads_per_block>>>(x_d, y_d, n);

    //Stop comp timing and sync
    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time,comp_start,comp_stop);


    hipMemcpy(y_h ,y_d, n*sizeof(int),hipMemcpyDeviceToHost);

    //Stop total timing and sync
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time,total_start,total_stop);

    for (i=n-15; i<n; i++)
		printf("%d\n", y_h[i]);

    /*
     * GPU timing
    */
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time(ms): %f\n", total_time-comp_time);

    hipFree(x_d);
    hipFree(y_d);

   return EXIT_SUCCESS;
}