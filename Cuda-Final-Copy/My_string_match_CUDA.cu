#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__
void MyPosition(char *buffer, char *pattern, long match_size, long pattern_size, int *total_matches, int *match)
{
	#include <stdio.h>
	//Deixnei poios ein o arithmos tu thread
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int i;

if (idx<match_size){
      for (i = 0; i < pattern_size && pattern[i] == buffer[i + idx]; ++i){
      		if (i >= pattern_size-1) {
         		match[idx] = 1;

						__syncthreads();
         		atomicAdd(total_matches,(long)1);
            }
   }}
}

int main (int argc, char *argv[]) {

	FILE *pFile;
	long file_size, match_size, pattern_size;
	int *total_matches_d;
	char * buffer_h, *buffer_d;
	char * filename, *pattern_h, *pattern_d;
	size_t result;
	int j, *match_h, *match_d;

        if (argc != 4) {
		printf ("Usage : %s <threads_per_block> <file_name> <string>\n", argv[0]);
		return 1;
        }
	filename = argv[2];
	pattern_h = argv[3];
  int threads_per_block = strtol(argv[1], NULL, 10);

	pFile = fopen ( filename , "rb" );
	if (pFile==NULL) {printf ("File error\n"); return 2;}

	// obtain file size:
	fseek (pFile , 0 , SEEK_END);
	file_size = ftell (pFile);
	rewind (pFile);
	printf("file size is %ld\n", file_size);

	// allocate memory to contain the file:
	buffer_h = (char*) malloc (sizeof(char)*file_size);
	if (buffer_h == NULL) {printf ("Memory error\n"); return 3;}

	// copy the file into the buffer:
	result = fread (buffer_h,1,file_size,pFile);
	if (result != file_size) {printf ("Reading error\n"); return 4;}

	pattern_size = strlen(pattern_h);
	match_size = file_size - pattern_size + 2;

	match_h = (int *) malloc (sizeof(int)*match_size);
	if (match_h == NULL) {printf ("Malloc error\n"); return 5;}

	int total_matches_h = 0;
	for (j = 0; j < match_size; j++){
		match_h[j]=0;
	}

	// MyShit

	float total_time, comp_time;
  hipEvent_t total_start, total_stop, comp_start, comp_stop;
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);
  hipEventCreate(&comp_start);
  hipEventCreate(&comp_stop);

	//Set Num of blocks
  int blocks = (match_size+threads_per_block-1)/threads_per_block;

	hipMalloc((void **)&buffer_d, file_size*sizeof(char));
	hipMalloc((void **)&pattern_d, pattern_size*sizeof(char));
	hipMalloc((void **)&total_matches_d,sizeof(long));
	hipMalloc((void **)&match_d,match_size*sizeof(int));


	// start total timing
  hipEventRecord(total_start);

	hipMemcpy(buffer_d, buffer_h, file_size*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(pattern_d, pattern_h, pattern_size*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(total_matches_d, &total_matches_h , sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(match_d,match_h,match_size*sizeof(int),hipMemcpyHostToDevice);

  //Start comp timing
  hipEventRecord(comp_start);

        /* Brute Force string matching */
	MyPosition<<<blocks,threads_per_block>>>(buffer_d,pattern_d,match_size,pattern_size,total_matches_d,match_d);

	//Stop comp timing and sync
	hipEventRecord(comp_stop);
  hipEventSynchronize(comp_stop);
  hipEventElapsedTime(&comp_time,comp_start,comp_stop);

	//Antigrafes stin mnimi
	hipMemcpy(&total_matches_h, total_matches_d, sizeof(long), hipMemcpyDeviceToHost);
	hipMemcpy(match_h,match_d,match_size*sizeof(int),hipMemcpyDeviceToHost);

	//Stop total timing and sync
  hipEventRecord(total_stop);
  hipEventSynchronize(total_stop);
  hipEventElapsedTime(&total_time,total_start,total_stop);

	//Typono kai opote vrisko match to proto stoixio kai tin thesi ston buffer
	for (j = 0; j < match_size; j++){
		if(match_h[j]==1){
			printf("[%c,%c,%d]",pattern_h[0],buffer_h[j],j);
		}
	}
        printf("\nTotal matches = %d\n", total_matches_h);

	/*
     * GPU timing
    */
  printf("Total time (ms): %f\n", total_time);
  printf("Kernel time (ms): %f\n", comp_time);
  printf("Data transfer time(ms): %f\n", total_time-comp_time);

	fclose (pFile);
	free (buffer_h);
	free (match_h);
	hipFree(buffer_d);
  hipFree(pattern_d);
  hipFree(total_matches_d);

	return EXIT_SUCCESS;
}