#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 128
#define base 0

__global__
void MyPosition(char *buffer, int *freq, int n)
{
  //Deixnei poios ein o arithmos tu thread
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  //printf("HALLO FROM THREAD : %d",idx);

if (idx<n){
      atomicAdd(&freq[buffer[idx]-base], 1);
   }
}

int main (int argc, char *argv[]) {

	FILE *pFile;
	long file_size;
	char * buffer_h,*buffer_d;
	char * filename;
	size_t result;
	int  j, freq_h[N],*freq_d;

        if (argc != 3) {
		printf ("Usage : %s <file_name> <threads_per_bock>\n", argv[0]);
		return 1;
        }
	filename = argv[1];
  int threads_per_block = strtol(argv[2], NULL, 10);
	pFile = fopen ( filename , "rb" );
	if (pFile==NULL) {printf ("File error\n"); return 2;}

	// obtain file size:
	fseek (pFile , 0 , SEEK_END);
	file_size = ftell (pFile);
	rewind (pFile);
	printf("file size is %ld\n", file_size);

	// allocate memory to contain the file:
	buffer_h = (char*) malloc (sizeof(char)*file_size);
	if (buffer_h == NULL) {printf ("Memory error\n"); return 3;}

	// copy the file into the buffer:
	result = fread (buffer_h,1,file_size,pFile);
	if (result != file_size) {printf ("Reading error\n"); return 4;}

	for (j=0; j<N; j++){
		freq_h[j]=0;
	}

  //MyShit---------------------

  float total_time, comp_time;
    hipEvent_t total_start, total_stop, comp_start, comp_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_stop);

  //Set Num of blocks
    int blocks = (file_size+threads_per_block-1)/threads_per_block;

    hipMalloc((void **)&buffer_d, file_size*sizeof(char));
    hipMalloc((void **)&freq_d, N*sizeof(int));

    // start total timing
    hipEventRecord(total_start);

    hipMemcpy(buffer_d, buffer_h, file_size*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(freq_d, freq_h, N*sizeof(int), hipMemcpyHostToDevice);

    //Start comp timing
    hipEventRecord(comp_start);

    MyPosition<<<blocks,threads_per_block>>>(buffer_d,freq_d,file_size);

    //Stop comp timing and sync
    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time,comp_start,comp_stop);

    hipMemcpy(freq_h, freq_d, N*sizeof(int), hipMemcpyDeviceToHost);

    //Stop total timing and sync
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time,total_start,total_stop);

  int total_elements=0;

	for (j=0; j<N; j++){
		printf("%d = %d\n", j+base, freq_h[j]);
    total_elements+=freq_h[j];
	}
  printf("Total elements are %d\n",total_elements);
   /*
     * GPU timing
    */
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time(ms): %f\n", total_time-comp_time);


    hipFree(buffer_d);
    hipFree(freq_d);

	fclose (pFile);
	free (buffer_h);

	return 0;
}